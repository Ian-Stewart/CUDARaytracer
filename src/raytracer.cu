#include "hip/hip_runtime.h"
//Ian Stewart & Alexander Newman
//CUDA/SDL ray tracer

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <X11/X.h>
#include <X11/Xlib.h>
#include <SDL/SDL.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "raystructs.h"
#include "raytracer.h"

#ifndef PI
#define PI           3.14159265358979323846
#endif
#define WIDTH 		1000
#define HEIGHT 		1000
#define DEPTH 		32
#define MAX_DEPTH	5

//__host__ __device__ indicates a function that is run on both the GPU and CPU
//__global__ indicates a CUDA kernel
__global__ void raytrace(Color3f *d_CUDA_Output, Sphere *d_spheres, Plane *d_planes, PointLight *d_lights, Camera *d_camera, int spherecount, int planecount, int lightcount, int w, int h, int c);//This actually does the raytracing

__host__ __device__ int sphereIntersect(Sphere *sphere, Ray *ray, HitRecord *hit, float tmin, float tmax);
__host__ __device__ int planeIntersect(Plane *plane, Ray *ray, HitRecord *hit, float tmin, float tmax);
__host__ __device__ int intersectScene(Sphere *d_spheres, Plane *d_planes, PointLight *d_lights, Ray *ray, HitRecord *hit, int spherecount, int planecount, int lightcount, float tmin, float tmax);
__host__ __device__ int triangleIntersect(Triangle *triangle, TriMesh *trimesh, Ray *ray, HitRecord *hit, float tmin, float tmax);

__host__ __device__ float VectorDot(Vector3f *v, Vector3f *u);
__host__ __device__ float findDeterminant(Vector3f *col0, Vector3f *col1, Vector3f *col2);

__host__ __device__ inline void InitVector(Vector3f *v, float ix, float iy, float iz);
__host__ __device__ inline void InitColor(Color3f *c, float ir, float ig, float ib);

__host__ __device__ void getShadingColor(Color3f *c, Sphere *d_spheres, Plane *d_planes, PointLight *d_lights, Ray *ray, HitRecord *hit, int spherecount, int planecount, int lightcount, int depth);
__host__ __device__ void getLight(PointLight *light, Vector3f *p, Vector3f *pos, Vector3f *lightDir, Color3f *c);
__host__ __device__ void getCameraRay(Ray *ray, Camera *d_camera, float x, float y);
__host__ __device__ void Refract(Vector3f *dir, Vector3f *normal, float ior, Vector3f *refr);
__host__ __device__ void Reflect(Vector3f *dir, Vector3f *normal, Vector3f *refl);
__host__ __device__ void VectorAdd(Vector3f *v, Vector3f *v1, Vector3f *v2);
__host__ __device__ void setNormalOfTriangle(Triangle *triangle);
__host__ __device__ void VectorSub(Vector3f *v, Vector3f *v1, Vector3f *v2);
__host__ __device__ void ScaleAdd(Vector3f *v0, float s, Vector3f *v1, Vector3f *v2);
__host__ __device__ void Normalize(Vector3f *v);
__host__ __device__ void Scale(Vector3f *v, float s);
__host__ __device__ void Negate(Vector3f *v);
__host__ __device__ void CrossProduct(Vector3f *out, Vector3f *v1, Vector3f *v2);
__host__ __device__ void PointOnRay(float t, Ray *ray, Vector3f *pos);

//Host only
void DrawScreen(SDL_Surface *screen);
void setpixel(SDL_Surface *screen, int x, int y, Uint8 r, Uint8 g, Uint8 b);
void initCamera(Camera *camera, Vector3f *in_eye, Vector3f *in_up, Vector3f *in_at, float in_fovy, float ratio);
unsigned int floatToUint(float f);

Camera camera;
int mouse_old_x;//Old mouse position
int mouse_old_y;
int width = WIDTH;
int height = HEIGHT;
void* d_CUDA_Output;//Device pointer for output
void* h_CUDA_Output;//Host pointer for output

int main(int argc, char *argv[]){
	dim3 threadsPerBlock(20,20);//Number of threads per block
	dim3 numBlocks(WIDTH/threadsPerBlock.x, HEIGHT/threadsPerBlock.y);

	h_CUDA_Output = malloc(sizeof(Color3f) * WIDTH * HEIGHT);//Allocate memory on host for output
	hipMalloc(&d_CUDA_Output, sizeof(Color3f) * WIDTH * HEIGHT);//Allocate memory on device for output
	//int i = 0;
	//hard-coded camera, for now
	Vector3f eye;
	Vector3f at;
	Vector3f up;
	InitVector(&eye, 0, -10, 0);
	InitVector(&at, 0, 0, 0);
	InitVector(&up, 1, 0, 0);
	initCamera(&camera, &eye, &up, &at, 40, 1);//Set up camera

	//hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
	
	SDL_Surface *screen;
	SDL_Event event;
	

	int c = 0;//For basic animation
	int keypress = 0;
	//int totalTris = 0;
	//int meshcount = 0;
	int spherecount = 3;
	int planecount = 1;
	int lightcount = 2;
	
	Sphere *spheres 	= (Sphere *)	malloc(sizeof(Sphere) * spherecount);//Scene will have three spheres
	//TriMesh *meshes 	= (TriMesh *)	malloc(sizeof(TriMesh) * 3);//Three trimeshes
	Plane *planes 		= (Plane *)	malloc(sizeof(Plane) * planecount);//One plane
	PointLight *lights 	= (PointLight *)malloc(sizeof(PointLight) * lightcount);//One light
	
	InitVector(&(spheres[0].center), -2, 0, 0);
	InitVector(&(spheres[1].center), 0, 0, 0);
	InitVector(&(spheres[2].center), 2, 0, 0);
	
	spheres[0].radius = 0.75;
	spheres[1].radius = 0.75;
	spheres[2].radius = 0.75;
	
	InitVector(&(planes[0].p), 0, 0, -2);
	InitVector(&(planes[0].normal), 0, 0, 1);
	Normalize(&(planes[0].normal));
	
	InitVector(&(lights[0].pos), 0, -10, 0);
	InitColor(&(lights[0].intensity), 25,25,25);
	InitVector(&(lights[1].pos), 0, 2, 4);
	InitColor(&(lights[1].intensity), 10,10,15);
	
	//Test material
	Material m;
	InitColor(&(m.Ka), 1, 1, 1);
	InitColor(&(m.Kd), 1, 1, 1);
	InitColor(&(m.Ks), 0.25, 0.25, 0.25);
	InitColor(&(m.Kr), 0, 0, 0);
	InitColor(&(m.Kt), 0, 0, 0);
	InitColor(&(m.Ie), 0, 0, 0);
	m.phong_exp = 10;
	m.ior = 0;
	
	spheres[0].material = m;
	spheres[1].material = m;
	spheres[2].material = m;
	
	planes[0].material = m;
	//End material
	
	//CUDA memory
	void* d_camera;
	//void* d_trimeshes;
	void* d_spheres;
	void* d_planes;
	//void* d_triangles;
	void* d_lights;
	
	//TriMesh *h_flattened_triangles;
	
	hipMalloc(&d_camera, sizeof(Camera));//Allocate memory for camera on host
	//hipMalloc(&d_trimeshes, sizeof(TriMesh) * meshcount);//Allocate memory for TriMesh structures
	hipMalloc(&d_spheres, sizeof(Sphere) * spherecount);//Allocate mem for spheres
	hipMalloc(&d_planes, sizeof(Plane) * planecount);
	hipMalloc(&d_lights, sizeof(PointLight) * lightcount);//For lights
	
	/*
	for(i = 0; i < meshcount; i++){
		totalTris += meshes[i].triangles;//Count up the total number of triangles
	}
	totalTris += meshcount * 12;//For bounding triangles
	*/
	
	//hipMalloc(&d_triangles, sizeof(Triangle) * totalTris);//Allocate space for flattened triangle data
	//h_flattened_triangles = (Triangle *)malloc(sizeof(Triangle) * totalTris);
	//Now dump triangle data to flat array.
	//Will be accessed by pointer shenanegans
	//Running 'offset' based on number of triangles in previous trimeshes
	//In this model, bounding volumes are the first 12 triangles of each trimesh segment
	//int offset = 0;
	//Triangle* currentptr;//Used in copying 
	//currentptr = h_flattened_triangles;
	//for(int i = 0; i < meshes; i++){
	//	memcpy(h_flattened_triangles, 
	//	currentptr += 12;//Increment pointer after copying bounding triangles
	//}
	
	//Finally, copy flattened data
	//hipMemcpy(d_triangles, h_flattened_triangles, sizeof(Triangle) * totalTris, hipMemcpyHostToDevice);

	//Begin copying from host to device
	//Copy camera
	hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
	//Copy planes
	hipMemcpy(d_planes, planes, sizeof(Plane) * planecount, hipMemcpyHostToDevice);
	//Copy spheres
	hipMemcpy(d_spheres, spheres, sizeof(Sphere) * spherecount, hipMemcpyHostToDevice);
	//Copy trimesh structs - cannot deep copy data
	//hipMemcpy(d_trimeshes, meshes, sizeof(Trimesh * meshcount, hipMemcpyHostToDevice);
	//Copy lights
	hipMemcpy(d_lights, lights, sizeof(PointLight) * lightcount, hipMemcpyHostToDevice);
	
	//End memory copying from host to device
	
	if(SDL_Init(SDL_INIT_VIDEO) < 0){
		return 1;
	}
	
	if(!(screen = SDL_SetVideoMode(width, height, DEPTH, SDL_HWSURFACE))){
		SDL_Quit();
		return 1;
	}
	timeval start, end;//For measuring frame length
	
	
	while(!keypress){
		printf("%i\n", c);
		gettimeofday(&start, NULL);
		//Launch Kernel
		raytrace<<<numBlocks, threadsPerBlock>>>(
			(Color3f *)d_CUDA_Output,
			(Sphere *) d_spheres,
			(Plane *) d_planes,
			(PointLight *) d_lights,
			(Camera *)d_camera,
			spherecount,
			planecount,
			lightcount,
			WIDTH,
			HEIGHT,
			c++
		);
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		hipDeviceSynchronize();//Wait for GPU to finish
		hipMemcpy(h_CUDA_Output, d_CUDA_Output, sizeof(Color3f) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);//Copy results of GPU kernel to host memory

		DrawScreen(screen);//Update the screen
		while(SDL_PollEvent(&event)){
			switch(event.type){
				case SDL_QUIT:
					keypress = 1;
					break;
				case SDL_KEYDOWN:
					keypress = 1;
					break;
			}//End switch(event.type)
		}//End while(SDL_PollEvent)
		gettimeofday(&end, NULL);
		printf("Frame took %lu msec\n", (end.tv_usec - start.tv_usec)/1000);
	}//End while(!keypress)
	SDL_Quit();
	return 0;
}

//Kernel that actually raytraces
//Size of each array of objects is given by 'x'count integers
__global__ void raytrace(
				Color3f *d_CUDA_Output,
				Sphere *d_spheres,
				Plane *d_planes,
				PointLight *d_lights,
				Camera *d_camera,
				int spherecount,
				int planecount,
				int lightcount,
				int w,
				int h,
				int c
			){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	Ray cameraRay;
	InitVector(&(cameraRay.d), 1, 0, 0);
	float x;
	float y; //(x,y) is the coordinate for this point in the image such that 0<=x,y<=1
	
	x = (float) i/ (float) w;
	y = (float) j/ (float) h;
	getCameraRay(&cameraRay, d_camera, x, y);//Get the camera ray
	
	float tmin = 0.001;
	float tmax = 1000;
	HitRecord hit;

	d_CUDA_Output[(j * w) + i].r = 0;
	d_CUDA_Output[(j * w) + i].g = 0;
	d_CUDA_Output[(j * w) + i].b = 0;
	
	if(intersectScene(d_spheres, d_planes, d_lights, &cameraRay, &hit, spherecount, planecount, lightcount, tmin, tmax) == 1){//Ray hit something in the scene
		Color3f c;
		InitColor(&c, 0, 0, 0);
		getShadingColor(&c, d_spheres, d_planes, d_lights, &cameraRay, &hit, spherecount, planecount, lightcount, 0);
		//Clamp to 1 - causes weird issues if this isn't done
		if(c.r > 1) c.r = 1;
		if(c.g > 1) c.g = 1;
		if(c.b > 1) c.b = 1;
		d_CUDA_Output[(j * w) + i] = c;
	}
}

//Given a ray and a scene, find the closest hiting point
__host__ __device__ int intersectScene(
					Sphere *d_spheres,
					Plane *d_planes,
					PointLight *d_lights,
					Ray *ray,
					HitRecord *hit,
					int spherecount,
					int planecount,
					int lightcount,
					float tmin,
					float tmax
				){
	int hitSomething = 0;//If ray intersects with no objects, return zero. Otherwise return 1.
	int i;//,j;
	//Check spheres
	for(i = 0; i < spherecount; i++){
		if(sphereIntersect(&(d_spheres[i]), ray, hit, tmin, tmax)  == 1){//ray intersects with sphere //TODO change &(spheres[i]) to spheres + i?
			hitSomething = 1;
			tmax = hit->t;//Reduce range, all hits after this must be closer to ray origin
		}
	}
	
	//Check triangle meshes
	//TODO: Check bounding volumes first to avoid checking every triangle needlessly
	/*
	for(i = 0; i < meshcount; i++){
		for(j = 0; j < meshes[i].triangles; j++){//Go through every triangle in the mesh
			if(triangleIntersect(&(meshes[i].data[j]), ray, hit, tmin, tmax) == 1){//ray intersects triangle
				hitSomething = 1;
				tmax = hit->t;
			}//end if
		}//end for(j = 0; j < scene->meshes[j].triangles...)
	}//End for(i = 0; i < scene->meshcount...)
	*/
	//Check planes
	for(i = 0; i < planecount; i++){//Check to see if ray intersects any planes
		if(planeIntersect(&(d_planes[i]), ray, hit, tmin, tmax) == 1){//ray intersects with plane
			hitSomething = 1;
			tmax = hit->t;
		}//endif
	}//end for (i = 0; i < scene->planecount...)
	
	return hitSomething;
}

//Get the shading color at a hitting point
//Recursively calls itself on reflective and refractive surfaces
__host__ __device__ void getShadingColor(Color3f *c, Sphere *d_spheres, Plane *d_planes, PointLight *d_lights, Ray *ray, HitRecord *hit, int spherecount, int planecount, int lightcount, int depth){
	InitColor(c, 0, 0, 0);
	Color3f lightColor;
	Vector3f lightPos, lightDir, flippedRay, R;
	Ray lightRay;
	HitRecord tempHit;
	float lightDist;
	int i;
	//Iterate through lights to find surface shading color
	for(i = 0; i < lightcount; i++){
		getLight(&(d_lights[i]), &(hit->pos), &lightPos, &lightDir, &lightColor);
		
		//Now check if shadowed
		lightDist = sqrtf((lightDir.x * lightDir.x) + (lightDir.y * lightDir.y) + (lightDir.z * lightDir.z));
		Normalize(&lightDir);
		
		lightRay.d = lightDir;
		lightRay.o = hit->pos;
		if(intersectScene(d_spheres, d_planes, d_lights, &lightRay, &tempHit, spherecount, planecount, lightcount, 0.01, lightDist) == 0){//No objects blocking the ray, do light calculation
			//Add diffuse portion
			c->r += lightColor.r * hit->material.Kd.r * fmaxf(VectorDot(&(hit->normal), &lightDir), 0);
			c->g += lightColor.g * hit->material.Kd.g * fmaxf(VectorDot(&(hit->normal), &lightDir), 0);
			c->b += lightColor.b * hit->material.Kd.b * fmaxf(VectorDot(&(hit->normal), &lightDir), 0);
			
			//Add specular portion
			//lightDir is the normalized vector from hit to light
			//lightPos is the position of lightColor
			//lightRay is the ray from hit to light
			
			Reflect(&(lightRay.d), &(hit->normal), &R);
			
			flippedRay = ray->d;
			Negate(&flippedRay);
			
			c->r += lightColor.r * hit->material.Ks.r * pow(fmaxf(0,VectorDot(&R, &flippedRay)),hit->material.phong_exp);
			c->g += lightColor.g * hit->material.Ks.g * pow(fmaxf(0,VectorDot(&R, &flippedRay)),hit->material.phong_exp);
			c->b += lightColor.b * hit->material.Ks.b * pow(fmaxf(0,VectorDot(&R, &flippedRay)),hit->material.phong_exp);
		}//end if(intersectScene() == 0)
	}//End light shading loop
	/*
	if(depth < MAX_DEPTH){
		Color3f reflectedColor, refractedColor;
		InitColor(&reflectedColor, 0, 0, 0);
		InitColor(&refractedColor, 0, 0, 0);
		Ray reflectedRay, refractedRay;
		HitRecord refractHit, reflectHit;
		
		//intersectScene(Scene *scene, Ray *ray, HitRecord *hit, float tmin, float tmax)
		
		//Find reflective portion
		if(hit->material.Kr.r > 0 || hit->material.Kr.g > 0 || hit->material.Kr.b > 0){//Surface is reflective
			reflectedRay.o = hit->pos;
			reflectedRay.d = ray->d;
			Negate(&(reflectedRay.d));
			Reflect(&(reflectedRay.d), &(hit->normal), &(reflectedRay.d));
				if(intersectScene(d_spheres, d_planes, d_lights, &reflectedRay, &reflectHit, spherecount, planecount,lightcount, 0.01, 1000) == 1){//reflected ray hits something
				getShadingColor(
					&reflectedColor, 
					d_spheres,
					d_planes,
					d_lights,
					&reflectedRay,
					&reflectHit,
					spherecount,
					planecount,
					lightcount,
					depth + 1
				);//Recursive call to get reflected color
					c->r += reflectHit.material.Kr.r * reflectedColor.r;
					c->g += reflectHit.material.Kr.g * reflectedColor.g;
					c->b += reflectHit.material.Kr.b * reflectedColor.b;
				}
		}//End if reflective
		
		//Find refractive portion
		if(hit->material.Kr.r > 0 || hit->material.Kr.r > 0 || hit->material.Kr.r > 0){//Material has refractive properties
			refractedRay.o = hit->pos;
			refractedRay.d = ray->d;
			Refract(&(ray->d), &(hit->normal), hit->material.ior, &(refractedRay.d));
			
			if(intersectScene(d_spheres, d_planes, d_lights, &refractedRay, &refractHit, spherecount, planecount,lightcount, 0.01, 1000) == 1){
				getShadingColor(
					&refractedColor, 
					d_spheres,
					d_planes,
					d_lights,
					&refractedRay,
					&refractHit,
					spherecount,
					planecount,
					lightcount,
					depth + 1
				);//Recursive call
				
				//Hack - refracted shading color is made more or less strong depending on the length of the ray through the objects
				//Seems to make stuff look really nice
				Vector3f vThroughObj = refractHit.pos;
				VectorSub(&vThroughObj, &vThroughObj, &(hit->pos));
				float factor = 1/(sqrtf((vThroughObj.x * vThroughObj.x) + (vThroughObj.y * vThroughObj.y) + (vThroughObj.z * vThroughObj.z)));
				if(factor > 1){
					c->r += hit->material.Kt.r * refractedColor.r;
					c->g += hit->material.Kt.g * refractedColor.g;
					c->b += hit->material.Kt.b * refractedColor.b;
				} else {
					c->r += hit->material.Kt.r * refractedColor.r * factor;
					c->g += hit->material.Kt.g * refractedColor.g * factor;
					c->b += hit->material.Kt.b * refractedColor.b * factor;
				}
			}//End if(intersectScene())
		}//End refractive section
		

		
	}*/
	//Add in emissive portion of material
	c->r += hit->material.Ie.r;
	c->g += hit->material.Ie.g;
	c->b += hit->material.Ie.b;
		//Add in ambient light portion
		//Not currently implemented
}

//Find the light intensity of a light at a point, and find useful information for shadow calculation
//Input: light, hit position
//Output: pos, lightDir, c
//Pos is the position of the light, lightDir is the direction from the light to p
__host__ __device__ void getLight(PointLight *light, Vector3f *p, Vector3f *pos, Vector3f *lightDir, Color3f *c){
	//light struct - pos, intensity
	*pos = light->pos;
	VectorSub(lightDir, pos, p);//lightDir = pos - p
	//Find light intensity
	//r = length of the vector from hit to light
	float r = 1/((lightDir->x * lightDir->x) + (lightDir->y * lightDir->y) + (lightDir->z * lightDir->z));
	*c = light->intensity;
	c->r = r * c->r;
	c->g = r * c->g;
	c->b = r * c->b;
	//Normalize(lightDir);
}

//Refract around a given normal and index of refraction
//Dir is assumed to be pointing into hit point
__host__ __device__ void Refract(Vector3f *dir, Vector3f *normal, float ior, Vector3f *refr){
	float mu;
	if(VectorDot(normal, dir) < 0){
		mu = 1/ior;
	} else {
		mu = ior;
	}
	
	float cos_thetai = VectorDot(dir, normal);
	float sin_thetai2 = 1 - (cos_thetai*cos_thetai);
	
	if(mu*mu*sin_thetai2 > 1){
		return;//Do nothing
	}
	
	float sin_thetar = mu*sqrtf(sin_thetai2);
	float cos_thetar = sqrtf(1 - (sin_thetar * sin_thetar));
	
	Vector3f out = *normal;
	
	if(cos_thetai > 0){
		Scale(&out, (-mu * cos_thetai) + cos_thetar);
		ScaleAdd(&out, mu, dir, &out);
	} else {
		Scale(&out, (-mu * cos_thetai) + cos_thetar);
		ScaleAdd(&out, mu, dir, &out);
	}
	
	Normalize(&out);
	*refr = out;
}

//Find a reflected ray given an incoming ray and a surface normal
//Assumes dir is pointing away from the hit point
__host__ __device__ void Reflect(Vector3f *dir, Vector3f *normal, Vector3f *refl){
	*refl = *normal;
	Scale(&(*refl), 2 * VectorDot(dir, normal));
	VectorSub(refl, refl, dir);
}

//Find the intersection of a ray and a triangle
__host__ __device__ int triangleIntersect(Triangle *triangle, TriMesh *trimesh, Ray *ray, HitRecord *hit, float tmin, float tmax){
	float a, b;//Barycentric alpha, beta
	
	Vector3f p2subp0 = triangle->p2;//p2-p0
	Vector3f p2subp1 = triangle->p2;//p2-p1
	Vector3f p2subo = triangle->p2;//p2-o
	VectorSub(&p2subp0, &p2subp0, &(triangle->p0));
	VectorSub(&p2subp1, &p2subp1, &(triangle->p1));
	VectorSub(&p2subo, &p2subo, &(ray->o));
	
	float detOfDenom;//Represents the common denominator in the cramer's rule determinant ({{a,b,c},{d,e,f},{g,h,i}})
	detOfDenom = findDeterminant(&(ray->d), &p2subp0, &p2subp1);
	
	if(detOfDenom == 0){//Ray is parallel to triangle
		return 0;
	}
	
	float t = findDeterminant(&p2subo, &p2subp0, &p2subp1)/detOfDenom;
	
	if(t > tmax || t < tmin){//t is out of bounds
		return 0;
	}
	
	a = findDeterminant(&(ray->d), &p2subo, &p2subp1)/detOfDenom;
	b = findDeterminant(&(ray->d), &p2subp0, &p2subo)/detOfDenom;
	
	if(a < 0 || b < 0 || a + b > 1){//Invalid barycentric coordinates - point is outside of triangle
		return 0;
	}
	
	//Now find coordinates of hit
	hit->t = t;
	hit->normal = triangle->n;
	
	Vector3f p0contrib = triangle->p0;
	Vector3f p1contrib = triangle->p1;
	Vector3f p2contrib = triangle->p2;
	
	Scale(&p0contrib, a);
	Scale(&p1contrib, b);
	Scale(&p2contrib, (1-a-b));
	
	hit->pos = p0contrib;
	VectorAdd(&(hit->pos), &(hit->pos), &p1contrib);
	VectorAdd(&(hit->pos), &(hit->pos), &p2contrib);
	
	//hit->material = triangle->material;
	hit->material = trimesh->material;
	return 1;
}

//Find the intersection of a sphere and a ray, if it exists
__host__ __device__ int sphereIntersect(Sphere *sphere, Ray *ray, HitRecord *hit, float tmin, float tmax){
	Vector3f v;
	InitVector(&v, 
		ray->o.x - sphere->center.x,
		ray->o.y - sphere->center.y,
		ray->o.z - sphere->center.z
	);
	float t;
	float B = 2*VectorDot(&v, &(ray->d));
	float C = VectorDot(&v, &v) - (sphere->radius * sphere->radius);
	float discriminant = sqrtf(B*B - 4*C);
	if(discriminant < 0){//Ray does not intersect sphere
		return 0;
	} else {
		float t1 = (-B + discriminant)/(2);
		float t2 = (-B - discriminant)/(2);
		if(t1 < tmin){
			t1 = t2;
		}
		if(t2 < tmin){
			t2 = t1;
		}
		//Now find smaller t
		if(t1 <= t2){
			t = t1;
		}
		if(t2 < t1){
			t = t2;
		}
		
		if(t > tmax || t < tmin){//Hit is out of bounds
			return 0;
		}
		
		hit->t = t;
		
		PointOnRay(hit->t, ray, &(hit->pos));//Find the hitting point and set hit->pos to it
		hit->material = sphere->material;//Set hit material
		//Normal at hitting point P is (P-Center)/|(P-Center) or (P-Center) normalized
		InitVector(&(hit->normal),
			hit->pos.x - sphere->center.x,
			hit->pos.y - sphere->center.y,
			hit->pos.z - sphere->center.z
		);
		Normalize(&(hit->normal));
		return 1;
	}//End else / if(discriminant < 0)
	
}

//Find the intersection of a ray and plane, if it exists
__host__ __device__ int planeIntersect(Plane *plane, Ray *ray, HitRecord *hit, float tmin, float tmax){
	Vector3f temp;
	temp = plane->p;
	VectorSub(&temp, &temp, &(ray->o));
	float denom = VectorDot(&(ray->d), &(plane->normal));
	if(denom == 0){//Ray is parallel to plane
		return 0;
	}
	float t = VectorDot(&temp, &(plane->normal)) / denom;
	if(t < tmin || t > tmax){//Hit is out of bounds
		return 0;
	}
	PointOnRay(t, ray, &(hit->pos));//Find the intersection point
	hit->t = t;
	hit->material = plane->material;//Set material of hit
	hit->normal = plane->normal;//Normal is always the same
	Normalize(&(hit->normal));//Should be normalized. Can't assume though...
	return 1;
}

//Used when setting up a trimesh. Given three points, finds the normal
__host__ __device__ void setNormalOfTriangle(Triangle *triangle){
	Vector3f v1;
	Vector3f v2;
	//v1 = p1 - p0
	v1.x = triangle->p1.x - triangle->p0.x;
	v1.x = triangle->p1.y - triangle->p0.y;
	v1.x = triangle->p1.z - triangle->p0.z;
	//v2 = p2 - p0
	v2.x = triangle->p2.x - triangle->p0.x;
	v2.x = triangle->p2.y - triangle->p0.y;
	v2.x = triangle->p2.z - triangle->p0.z;
	CrossProduct(&(triangle->n), &v1, &v2);
	Normalize(&(triangle->n));
}

//Given three columns representing a matrix, finds the determinant
__host__ __device__ float findDeterminant(Vector3f *col0, Vector3f *col1, Vector3f *col2){
	return 
	(col0->x*(col1->y*col2->z - col1->z*col2->y)) 
	- (col1->x*(col0->y*col2->z - col0->z*col2->y))
	+ (col2->x*(col0->y*col1->z - col0->z*col1->y));
}

//Set up camera rays for ray tracer
__host__ __device__ void getCameraRay(Ray *ray, Camera *d_camera, float x, float y){
	Vector3f direction;
	InitVector(&direction, 0, 0, 0);
	ScaleAdd(&direction, x, &(d_camera->across), &(d_camera->corner));
	ScaleAdd(&direction, y, &(d_camera->up), &direction);
	VectorSub(&direction, &direction, &(d_camera->center));
	Normalize(&direction);
	ray->o = d_camera->center;
	ray->d = direction;
}

//Find a point on a ray given some t and a ray and load that point into pos
__host__ __device__ void PointOnRay(float t, Ray *ray, Vector3f *pos){
	pos->x = ray->o.x + (ray->d.x*t);
	pos->y = ray->o.y + (ray->d.y*t);
	pos->z = ray->o.z + (ray->d.z*t);
}

//Find the dot product of a vector
__host__ __device__ float VectorDot(Vector3f *v, Vector3f *u){
	return (v->x * u->x) + (v->y * u->y) + (v->z * u->z);
}

//Compute the cross product of a vector
//v1 x v2 = |{{i,j,k},{v1.x,v1.y,v1.z},{v2.x,v2.y,v2.z}}|
__host__ __device__ void CrossProduct(Vector3f *out, Vector3f *v1, Vector3f *v2){
	out->x = (v1->y * v2->z) - (v1->z * v2->y);
	out->y = -(v1->x * v2->z) - (v1->z * v2->x);
	out->z = (v1->x * v2->y) - (v1->y * v2->x);
}

//Negates a vector v = -v
__host__ __device__ void Negate(Vector3f *v){
	v->x = -(v->x);
	v->y = -(v->y);
	v->z = -(v->z);
}

//Scales a vector v = s*v
__host__ __device__ void Scale(Vector3f *v, float s){
	v->x = s*(v->x);
	v->y = s*(v->y);
	v->z = s*(v->z);
}

//v = v1 + v2
__host__ __device__ void VectorAdd(Vector3f *v, Vector3f *v1, Vector3f *v2){
	v->x = v1->x + v2->x;
	v->y = v1->y + v2->y;
	v->z = v1->z + v2->z;
}

//Normalizes a vector (sets v = v/|v|)
__host__ __device__ void Normalize(Vector3f *v){
	float magnitude = sqrtf( pow(v->x,2) + pow(v->y,2) + pow(v->z,2) );//Length of vector v
	v->x = (v->x)/magnitude;
	v->y = (v->y)/magnitude;
	v->z = (v->z)/magnitude;
}

//v = v1-v2
__host__ __device__ void VectorSub(Vector3f *v, Vector3f *v1, Vector3f *v2){
	v->x = (v1->x) - (v2->x);
	v->y = (v1->y) - (v2->y);
	v->z = (v1->z) - (v2->z);
}

//Sets a vector to some value
__host__ __device__ void InitVector(Vector3f *v, float ix, float iy, float iz){
	v->x = ix;
	v->y = iy;
	v->z = iz;
}

//Sets a color to some value
__host__ __device__ void InitColor(Color3f *c, float ir, float ig, float ib){
	c->r = ir;
	c->g = ig;
	c->b = ib;
}

//scaleadd v0 = s*v1 + v2
__host__ __device__ void ScaleAdd(Vector3f *v0, float s, Vector3f *v1, Vector3f *v2){
	Vector3f v;
	v.x = s*(v1->x);
	v.y = s*(v1->y);
	v.z = s*(v1->z);
	
	v.x += v2->x;
	v.y += v2->y;
	v.z += v2->z;
	
	v0->x = v.x;
	v0->y = v.y;
	v0->z = v.z;
}

void initCamera(Camera *camera, Vector3f *in_eye, Vector3f *in_up, Vector3f *in_at, float in_fovy, float ratio){
	//Update camera information
	camera->eye = *in_eye;
	camera->up = *in_up;
	camera->at = *in_at;
	camera->fovy = in_fovy;
	camera->aspect_ratio = ratio;
	
	//Compute points of image plane
	float dist = 1;
	float top = dist * (tanf((camera->fovy * PI)/360));
	float bottom = -top;
	float right = ratio*top;
	float left = -right;
	
	Vector3f gaze;
	VectorSub(&gaze, &(camera->at), &(camera->eye));//gaze = at-eye
	
	camera->center = camera->eye;
	Vector3f W = gaze;
	Negate(&W);
	Normalize(&W);
	Vector3f V = camera->up;
	Vector3f U;
	InitVector(&U, 0, 0, 0);
	CrossProduct(&U, &V, &W);//U = VxW
	Normalize(&U);
	CrossProduct(&V, &W, &U);
	
	InitVector(&(camera->corner), 0, 0, 0);
	ScaleAdd(&(camera->corner), left, &U, &(camera->center));
	ScaleAdd(&(camera->corner), bottom, &V, &(camera->corner));
	ScaleAdd(&(camera->corner), -dist, &W, &(camera->corner));
	
	camera->across = U;
	Scale(&(camera->across), right-left);
	
	camera->up = V;
	Scale(&(camera->up), top-bottom);
}

//Converts float 0-1 to 0-255
unsigned int floatToUint(float f){
	unsigned int u = (int)(f*255);
	return u;
}

//Draws the output of the CUDA kernel on the screen
void DrawScreen(SDL_Surface *screen){
	int y = 0;
	int x = 0;
	if(SDL_MUSTLOCK(screen)){
		if(SDL_LockSurface(screen)){
			return;
		}
	}
	
	Color3f *cudaout = (Color3f *)h_CUDA_Output;
	
	for(y = 0; y < HEIGHT;y++){
		for(x = 0; x < WIDTH;x++){
			setpixel(screen, x, y, floatToUint(cudaout[(x * WIDTH) + y].r), floatToUint(cudaout[(x * WIDTH) + y].g), floatToUint(cudaout[(x * WIDTH) + y].b));
		}
	}//End for(y..){
		
	if(SDL_MUSTLOCK(screen)){
		SDL_UnlockSurface(screen);
	}
	SDL_Flip(screen);
}

void setpixel(SDL_Surface *screen, int x, int iny, Uint8 r, Uint8 g, Uint8 b){
	Uint32 *pixmem32;
	Uint32 colour;  
	int y = iny*HEIGHT;
	colour = SDL_MapRGB( screen->format, r, g, b );

	pixmem32 = (Uint32*) screen->pixels  + y + x;
	*pixmem32 = colour;
}